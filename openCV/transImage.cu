#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <math.h>
#include <hip/hip_runtime.h>
#define RED = 2
#define GREEN = 1
#define BLUE = 0
#define chanDepth = 3

using namespace cv;

__global__ void matMul(double *image, double *resImage,const size_t& rows,const size_t& cols){
	/* it will multiply each pixel of given image per 2 */
	int ti = blockIdx.y*blockDim.y+threadIdx.y;
	int tj = blockIdx.x*blockDim.x+threadIdx.x;
	if(ti < rows && tj < cols){
		for(size_t k=0; k<rows; k++){
			resImage[(ti*rows + tj)*chanDepth + RED] *= 2;
			resImage[(ti*rows + tj)*chanDepth + GREEN] *= 2;
			resImage[(ti*rows + tj)*chanDepth + BLUE] *= 2;
		}	
	}
}

int main(int argc, char** argv ){
	if(argc != 2){
		printf("usage: DisplayImage.out <Image_Path>\n");
		return -1;
	}

	Mat image;
	image = imread(argv[1],1);

	if(!image.data){
		printf("No image data \n");
		return -1;
	}
	unsigned char *h_rawImage, *d_rawImage, *h_procImage, *d_procImage;

	/* Memory management */
	Size imgSize = image.size();
	size_t imgHeight, imgWidth;
	imgHeight = imgSize.height;
	imgWidth = imgSize.width;
	size_t reqMem = imgHeight*imgWidth*image.channels()*sizeof(unsigned char);
	h_rawImage = (unsigned char *)malloc(reqMem);
	h_procImage = (unsigned char *)malloc(reqMem);
	h_rawImage = image.data;	
	hipMalloc((void**)&d_rawImage,reqMem);
	hipMalloc((void**)&d_procImage,reqMem);	
	dim3 blockSize(32,32,1);
	size_t reqBlocks = ceil((double)reqMem/1024);
	size_t blocksInX = ceil(sqrt(reqBlocks));
	size_t blocksInY = blocksInX;
	dim3 gridSize(blocksInX,blocksInY,1);

	/* Transfering data to device */
	hipMemcpy(d_rawImage,h_rawImage,reqMem,hipMemcpyHostToDevice);
	/* Operating */	
	matMul<<<gridSize,blockSize>>>(d_rawImage,d_procImage,imgHeight,imgWidth);		
	/* Recovering data to host */
	hipMemcpy(h_procImage,d_procImage,reqMem,hipMemcpyDeviceToHost);

	/* Saving Image */
	Mat procImage;
	procImage.create(imgHeight,imgWidth,CV_8UC3);	
	procImage.data = h_procImage;
	imwrite("output.jpg",procImage);

	/* Freeing memory */
	hipFree(d_rawImage);
	hipFree(d_procImage);
	free(h_rawImage);
	free(h_procImage);
	return 0;
}

