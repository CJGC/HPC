#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <math.h>
#include <hip/hip_runtime.h>
#define RED 2
#define GREEN 1
#define BLUE 0
#define chanDepth 3

using namespace cv;

__host__ void checkCudaState(hipError_t& cudaState,const char *message,unsigned char *h_grayScale){
   /* it will print an error message if there is */
   if(cudaState != hipSuccess){
      printf("%s",message);
      free(h_grayScale);
      exit(-1);
   }
}

__global__ void grayScale(unsigned char *image,unsigned char *resImage,int rows,int cols){
   /* it will turn an image to gray scale image */
   int ti = blockIdx.y*blockDim.y+threadIdx.y;
   int tj = blockIdx.x*blockDim.x+threadIdx.x;
   if(ti < rows && tj < cols){
      int pos = (ti*cols + tj)*chanDepth;
      resImage[ti*cols + tj] = image[pos+BLUE]*0.07 + image[pos+GREEN]*0.72 + image[pos+RED]*0.21;
   }
}

int main(int argc, char** argv ){
   if(argc != 2){
      printf("usage: DisplayImage.out <Image_Path>\n");
      return -1;
   }

   Mat image;
   image = imread(argv[1],1);
   hipError_t cudaState = hipSuccess;

   if(!image.data){
      printf("No image data \n");
      return -1;
   }
   unsigned char *h_rawImage, *d_rawImage,*h_grayScale, *d_grayScale;

   /* Memory management */
   Size imgSize = image.size();
   int imgHeight = imgSize.height, imgWidth = imgSize.width;
   int reqMemForRawImg = imgHeight*imgWidth*image.channels()*sizeof(unsigned char);
   int reqMemForGrayScaImg = imgHeight*imgWidth*sizeof(unsigned char);
   h_grayScale = (unsigned char *)malloc(reqMemForGrayScaImg);
   cudaState = hipMalloc((void**)&d_rawImage,reqMemForRawImg);
   checkCudaState(cudaState,"Was not possible allocate memory for d_rawImage\n",h_grayScale);
   cudaState = hipMalloc((void**)&d_grayScale,reqMemForGrayScaImg);	
   checkCudaState(cudaState,"Was not possible allocate memory for d_grayScale\n",h_grayScale);
   h_rawImage = image.data; 
   dim3 blockSize(32,32,1);
   int reqBlocksInX = ceil((double)imgHeight/32.0); 
   int reqBlocksInY = ceil((double)imgWidth/32.0); 
   dim3 gridSize(reqBlocksInY,reqBlocksInX,1);
 
   /* Transfering data to device */
   cudaState = hipMemcpy(d_rawImage,h_rawImage,reqMemForRawImg,hipMemcpyHostToDevice);
   checkCudaState(cudaState,"Was not possible copy data from h_rawImage to d_rawImage\n",h_grayScale);
   /* Operating */	
   grayScale<<<gridSize,blockSize>>>(d_rawImage,d_grayScale,imgHeight,imgWidth);
   hipDeviceSynchronize();
   /* Recovering data to host */
   cudaState = hipMemcpy(h_grayScale,d_grayScale,reqMemForGrayScaImg,hipMemcpyDeviceToHost);
   checkCudaState(cudaState,"Was not possible copy data from d_grayScale to h_grayScale\n",h_grayScale);
   /* Saving Image */
   Mat procImage;
   procImage.create(imgHeight,imgWidth,CV_8UC1);	
   procImage.data = h_grayScale;
   imwrite("output.jpg",procImage);

   /* Freeing memory */
   hipFree(d_rawImage);
   hipFree(d_grayScale);
   // h_rawImage is a pointer to Mat's buffer, when Mat's buffer is  destroyed 
   // memory is freed
   free(h_grayScale);
}

