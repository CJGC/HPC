
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int tx = blockIdx.x*blockDim.x+threadIdx.x;
    int ty = blockIdx.y*blockDim.y+threadIdx.y;

    // Make sure we do not go out of bounds
    if (tx < n && ty < n){
      int k=0; double data=0.0;
      for(k;k<n;k++){
        data += a[ty*n+k]*b[k*n+tx];
      }
      c[ty*n+tx] = data;
    }
}

int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 100000;

    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;

    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);

    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc((void **)&d_a, bytes);
    hipMalloc((void **)&d_b, bytes);
    hipMalloc((void **)&d_c, bytes);

    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }

    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    //int blockSize, gridSize;

    // Number of threads in each thread matrix block
    //blockSize = 1024;
    dim3 dimBlock(32,32,1);

    // Number of thread blocks in matrix grid
    dim3 dimGrid(32,32,1);
    //gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    vecAdd<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, n);

    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);

    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
