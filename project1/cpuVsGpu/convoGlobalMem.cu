#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define RED 2
#define GREEN 1
#define BLUE 0
#define chanDepth 3

using namespace cv;

__host__ void checkCudaState(hipError_t& cudaState,const char *message){
   /* it will print an error message if there is */
   if(cudaState != hipSuccess) printf("%s",message);
}

__device__ uchar clamp(int value){
    if(value < 0) return 0;
    if(value > 255) return 255;
    return (uchar)value;
}

__global__ void sobeFilt(uchar *image,uchar *resImage,int width,int height,char *mask){
    uint row = blockIdx.y*blockDim.y+threadIdx.y;
    uint col = blockIdx.x*blockDim.x+threadIdx.x;
    uint maskWidth = 3;//sqrt((double)sizeof(mask)/sizeof(char));
    int Pvalue = 0;
    int stPointRow = row - (maskWidth/2); //start point with respect mask
    int stPointCol = col - (maskWidth/2); //start point with respect mask

    if(row < height && col < width){
      for(int i=0; i<maskWidth; i++){
          int iMask = stPointRow + i;
          for(int j=0; j<maskWidth; j++){
              int jMask = stPointCol + j;
              if((iMask >=0 && iMask < height)&&(jMask >=0 && jMask < width))
                  Pvalue += image[(iMask*width) + jMask] * mask[i*maskWidth+j];
          }
      }
      resImage[row*width+col] = clamp(Pvalue);
    }
}

__global__ void grayScale(uchar *image,uchar *resImage,int rows,int cols){
   /* it will turn an image to gray scale image */
   int ti = blockIdx.y*blockDim.y+threadIdx.y;
   int tj = blockIdx.x*blockDim.x+threadIdx.x;
   if(ti < rows && tj < cols){
      int pos = (ti*cols + tj)*chanDepth;
      resImage[ti*cols + tj] = image[pos+BLUE]*0.07 + image[pos+GREEN]*0.72 + image[pos+RED]*0.21;
   }
}

int main(int argc, char** argv ){
   if(argc != 3){
      printf("usage: %s <image> <numCases>\n",argv[0]);
      return -1;
   }

   Mat image;
   image = imread(argv[1],1);
   hipError_t cudaState = hipSuccess;

   if(!image.data){
      printf("No image data \n");
      return -1;
   }

   int cases = atoi(argv[2]);
   FILE *data= fopen("data.txt","w+");
   do{
     clock_t start, end;
     double usedTime = 0.0;
     /* Memory data management */
     Size imgSize = image.size();
     int imgHeight = imgSize.height, imgWidth = imgSize.width;
     int reqMemForRawImg = imgHeight*imgWidth*image.channels()*sizeof(uchar);
     int reqMemForProcImg = imgHeight*imgWidth*sizeof(uchar);
     uchar *h_rawImage = NULL, *h_grayScale = NULL, *h_sobelImage = NULL;
     uchar *d_rawImage = NULL, *d_grayScale = NULL, *d_sobelImage = NULL;
     char h_mask[] = {-1,0,1,-2,0,2,-1,0,1}, *d_mask=NULL;
     uint maskSize = sizeof(h_mask);

     h_grayScale = (uchar *)malloc(reqMemForProcImg);
     h_sobelImage = (uchar *)malloc(reqMemForProcImg);

     cudaState = hipMalloc((void**)&d_rawImage,reqMemForRawImg);
     checkCudaState(cudaState,"Unallocated memory for d_rawImage\n");
     cudaState = hipMalloc((void**)&d_grayScale,reqMemForProcImg);
     checkCudaState(cudaState,"Unallocated memory for d_grayScale\n");
     cudaState = hipMalloc((void**)&d_sobelImage,reqMemForProcImg);
     checkCudaState(cudaState,"Unallocated memory for d_sobelImage\n");
     cudaState = hipMalloc((void**)&d_mask,maskSize);
     checkCudaState(cudaState,"Unallocated memory for d_mask\n");

     if(d_rawImage != NULL && d_grayScale != NULL && d_sobelImage != NULL && d_mask != NULL){
       /* Setting kernel properties */
       h_rawImage = image.data;
       dim3 blockSize(32,32,1);
       int reqBlocksInX = ceil((double)imgHeight/32.0);
       int reqBlocksInY = ceil((double)imgWidth/32.0);
       dim3 gridSize(reqBlocksInY,reqBlocksInX,1);

       start = clock();
       /* Transfering and processing data to obtain grayimage */
       cudaState = hipMemcpy(d_rawImage,h_rawImage,reqMemForRawImg,hipMemcpyHostToDevice);
       checkCudaState(cudaState,"Impossible copy data from h_rawImage to d_rawImage\n");
       grayScale<<<gridSize,blockSize>>>(d_rawImage,d_grayScale,imgHeight,imgWidth);
       hipDeviceSynchronize();
       /* Transfering and processing data to obtain sobel image */
       cudaState = hipMemcpy(d_mask,h_mask,maskSize,hipMemcpyHostToDevice);
       checkCudaState(cudaState,"Impossible copy data from mask to d_mask\n");
       sobeFilt<<<gridSize,blockSize>>>(d_grayScale,d_sobelImage,imgWidth,imgHeight,d_mask);
       hipDeviceSynchronize();

       /* Recovering data of grayScale image to h_grayScale */
       cudaState = hipMemcpy(h_grayScale,d_grayScale,reqMemForProcImg,hipMemcpyDeviceToHost);
       checkCudaState(cudaState,"Impossible copy data from d_grayScale to h_grayScale\n");
       /* Recovering data of sobelImage to h_sobelImage */
       cudaState = hipMemcpy(h_sobelImage,d_sobelImage,reqMemForProcImg,hipMemcpyDeviceToHost);
       checkCudaState(cudaState,"Impossible copy data from d_sobelImage to h_sobelImage\n");
       end = clock();
       usedTime = ((double)(end - start))/ CLOCKS_PER_SEC;
       fprintf(data,"%f,",usedTime);

       /* Processing data with host using opencv */
       start = clock();
       Mat grayscale_opencv, gradient_x, abs_gradient_x;
       cvtColor(image, grayscale_opencv, CV_BGR2GRAY);
       Sobel(grayscale_opencv,gradient_x,CV_8UC1,1,0,3,1,0,BORDER_DEFAULT);
       convertScaleAbs(gradient_x, abs_gradient_x);
       end = clock();
       usedTime = ((double)(end - start))/ CLOCKS_PER_SEC;
       fprintf(data,"%f\n",usedTime);
       imwrite("grayscale_opencv.jpg",grayscale_opencv);
       imwrite("sobel_opencv.jpg",abs_gradient_x);

       /* Saving Images */
       Mat grayscaleImage, sobelImage;
       grayscaleImage.create(imgHeight,imgWidth,CV_8UC1);
       sobelImage.create(imgHeight,imgWidth,CV_8UC1);
       grayscaleImage.data = h_grayScale;
       sobelImage.data = h_sobelImage;
       imwrite("grayscale.jpg",grayscaleImage);
       imwrite("sobel.jpg",sobelImage);
     }

     /* Freeing device's memory */
     if(d_rawImage != NULL) hipFree(d_rawImage);
     if(d_grayScale != NULL) hipFree(d_grayScale);
     if(d_sobelImage != NULL) hipFree(d_sobelImage);
     if(d_mask != NULL) hipFree(d_mask);

     /* Freeing host's memory */
     // h_rawImage is a pointer to Mat's buffer, when Mat's buffer is  destroyed
     // memory is freed
     if(h_grayScale != NULL) free(h_grayScale);
     if(h_sobelImage != NULL) free(h_sobelImage);
     cases--;
   }while(cases > 0);
   fclose(data);
}
